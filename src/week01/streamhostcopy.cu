#include "hip/hip_runtime.h"

#include "common.h"

// CUDA内核函数定义
__global__ void simpleKernel(int *data, int val) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] += val;
}

int cuda_stream_host_copy() {
    const int numElements = 1024;
    const int size = numElements * sizeof(int);
    const int numStreams = 5;
    int *d_data;

    // 分配设备内存
    hipMalloc(&d_data, size);

    // 初始化设备内存
    hipMemset(d_data, 0, size);

    // 创建5个CUDA流
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // 在每个流上执行内核
    for (int i = 0; i < numStreams; ++i) {
        simpleKernel<<<1, 1024, 0, streams[i]>>>(d_data, i);
    }

    // 同步所有流
    for (int i = 0; i < numStreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    // 销毁所有流
    for (int i = 0; i < numStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    // 清理设备内存
    hipFree(d_data);

    return 0;
}
