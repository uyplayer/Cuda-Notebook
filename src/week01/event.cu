#include "hip/hip_runtime.h"



#include "common.h"



__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    auto i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}


bool event_cuda(){
    const auto numElements = 50000;
    auto size = numElements * sizeof(float);

    std::vector<float> h_A(numElements);
    std::vector<float> h_B(numElements);
    std::vector<float> h_C(numElements);


    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);

    for (int i = 0; i < numElements; ++i) {
        h_A[i] = static_cast<float>(dis(gen));
        h_B[i] = static_cast<float>(dis(gen));
    }

    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;

    // 分配设备端内存
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 复制数据从主机到设备端
    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);


    // 创建事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 开始就记录
    hipEventRecord(start);

    // 启动核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // 记录结束了
    hipEventRecord(stop);
    hipEventSynchronize(stop);


    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Vector Add took " << milliseconds << " milliseconds\n";

    // 销毁事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 复制结果从设备到主机端内存
    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // 验证结果
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    // 清理
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "Test PASSED\n";

    // 重置设备
    hipDeviceReset();

    return true;

}