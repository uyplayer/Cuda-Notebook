//
// Created by uyplayer on 2024/5/20.
//


#include "error.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>


//  Fortran风格的1-based索引和列主存储
// row ,col ,leading dimension(number of rows
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
// C风格的0-based索引和行主存储
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// row and col
#define M 6
#define N 5


// cuBLAS: 0-based indexing
static __inline__ void modify0(hipblasHandle_t handle, float* m, int ldm, int n, int p, int q, float alpha, float beta)
{
    hipblasSscal(handle, n - q + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
    hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}


// cuBLAS: 1-based indexing
void base_1()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;


    float* devPtrA;
    float* a = 0;
    a = (float*)malloc(M * N * sizeof (*a));
    if (a == NULL)
    {
        std::cout << "host memory allocation failed" << std::endl;
        exit(1);
    }
    for (int j = 1; j <= N; j++)
    {
        for (int i = 1; i <= M; i++)
        {
            a[IDX2F(i, j, M)] = (float)(i + (j - 1) * M);
        }
    }

    cudaStat = hipMalloc((void**)&devPtrA, M * N * sizeof(*a));
    if (cudaStat != hipSuccess)
    {
        std::cout << "device memory allocation failed" << std::endl;
        free(a);
        exit(1);
    }
    // initialize handle
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed" << std::endl;
        hipFree(devPtrA);
        free(a);
        exit(1);
    }
    // copy the array to the device
    stat = hipblasSetMatrix(M,N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("data download failed");
        free(a);
        hipFree(devPtrA);
        hipblasDestroy(handle);
        exit(1);
    }
}

// cuBLAS: 0-based indexing
void base_0()
{
}
