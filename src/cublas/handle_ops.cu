//
// Created by uyplayer on 2024/5/21.
//


#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>


void des_handle()
{
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "Failed to create handle" << std::endl;
        exit(1);
    }
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS destruction failed\n");
        std::cout << "Failed to destroy handle" << std::endl;
        exit(1);
    }
}


void addVector()
{
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);

    int m = 2, n = 2, k = 2;
    float alpha = 1.0f;
    float beta = 0.0f;

    float h_A[] = {1.0, 2.0, 3.0, 4.0};
    float h_B[] = {5.0, 6.0, 7.0, 8.0};
    float h_C[] = {0.0, 0.0, 0.0, 0.0};

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    // 执行矩阵乘法：C = alpha * A * B + beta * C
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);

    // 复制结果回主机
    hipblasGetMatrix(m, n, sizeof(float), d_C, m, h_C, m);

    // 打印结果
    printf("Result matrix C:\n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%f ", h_C[i * n + j]);
        }
        printf("\n");
    }

    // 释放资源
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}


void checkCublasStatus()
{
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        switch (status)
        {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("CUBLAS_STATUS_NOT_INITIALIZED\n");
            break;
        case HIPBLAS_STATUS_ALLOC_FAILED:
            printf("CUBLAS_STATUS_ALLOC_FAILED\n");
            break;
        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("CUBLAS_STATUS_INVALID_VALUE\n");
            break;
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            printf("CUBLAS_STATUS_ARCH_MISMATCH\n");
            break;
        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("CUBLAS_STATUS_MAPPING_ERROR\n");
            break;
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            printf("CUBLAS_STATUS_EXECUTION_FAILED\n");
            break;
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            printf("CUBLAS_STATUS_INTERNAL_ERROR\n");
            break;
        default:
            printf("Unknown error\n");
        }
    }
    hipblasDestroy(handle);
}
