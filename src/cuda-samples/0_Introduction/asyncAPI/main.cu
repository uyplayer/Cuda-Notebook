#include "hip/hip_runtime.h"
//
// Created by uyplayer on 2024-06-13.
//
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cxxopts.hpp"
#include <error.h>

__global__ void increment_kernel(int *g_data, int inc_value,int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        g_data[idx] = g_data[idx] + inc_value;
    }
}


int main(int argc, char **argv) {
    std::cout << "Hello, asyncAPI!" << std::endl;

    // 获取设备ID
    cxxopts::Options options("asyncAPI", "A brief description");
    options.add_options()
        ("d,device", "Device ID", cxxopts::value<int>()->default_value("0"));
    const auto command_result = options.parse(argc, argv);
    int device_id = command_result["device"].as<int>();
    std::cout << "Device ID: " << device_id << std::endl;

    constexpr int n = 16 * 1024 * 1024;
    constexpr int nbytes = n * sizeof(int);
    int *h_a, *d_b;

    HANDLE_ERROR(hipHostMalloc(&h_a, nbytes));
    HANDLE_ERROR(hipMemset(h_a, 0, nbytes));
    HANDLE_ERROR(hipMalloc(&d_b, nbytes));
    HANDLE_ERROR(hipMemset(d_b, 255, nbytes));

    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3((n + threads.x - 1) / threads.x, 1);

    hipStream_t stream;
    HANDLE_ERROR(hipStreamCreate(&stream));

    // 创建 CUDA 事件
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float gpu_time = 0.0f;

    HANDLE_ERROR(hipProfilerStart());
    HANDLE_ERROR(hipEventRecord(start, stream));
    HANDLE_ERROR(hipMemcpyAsync(d_b, h_a, nbytes, hipMemcpyHostToDevice, stream));
    increment_kernel<<<blocks, threads, 0, stream>>>(d_b, 963, n);
    HANDLE_ERROR(hipMemcpyAsync(h_a, d_b, nbytes, hipMemcpyDeviceToHost, stream));
    HANDLE_ERROR(hipEventRecord(stop, stream));
    HANDLE_ERROR(hipProfilerStop());
    // HANDLE_ERROR(hipStreamSynchronize(stream));
    unsigned long int counter = 0;
    while (hipEventQuery(stop) == hipErrorNotReady) {
        std::cout << "Waiting for GPU to finish... " << counter << std::endl;
        counter++;
    }
    HANDLE_ERROR(hipEventElapsedTime(&gpu_time, start, stop));
    std::cout << "Time spent on GPU: " << gpu_time << " ms" << std::endl;

    HANDLE_ERROR(hipHostFree(h_a));
    HANDLE_ERROR(hipFree(d_b));
    HANDLE_ERROR(hipStreamDestroy(stream));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    std::cout << "Completed successfully!" << std::endl;
    return 0;
}