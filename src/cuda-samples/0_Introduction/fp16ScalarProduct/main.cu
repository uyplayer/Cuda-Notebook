//
// Created by uyplayer on 2024-06-19.
//




#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "error.h"


#define NUM_OF_BLOCKS 128
#define NUM_OF_THREADS 256


void generateInput(half2 *a, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        half2 temp;
        temp.x = static_cast<float>(rand() % 4);
        temp.y = static_cast<float>(rand() % 2);
        a[i] = temp;
    }
}


int main() {
    std::cout << "Hello, fp16ScalarProduct !" << std::endl;

    // 使用BF16 数据类型
    size_t size = NUM_OF_BLOCKS * NUM_OF_THREADS * 16;
    half2 *vec[2];
    half2 *devVec[2];

    float *results;
    float *devResults;

    int device = 0;
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, device));
    HANDLE_ERROR(hipSetDevice(device));

    for (int i = 0; i < 2; ++i) {
        HANDLE_ERROR(hipHostMalloc((void **) &vec[i], size * sizeof *vec[i]));
        HANDLE_ERROR(hipMalloc((void **) &devVec[i], size * sizeof *devVec[i]));

    }

    HANDLE_ERROR(hipHostMalloc((void **) &results, NUM_OF_BLOCKS * sizeof *results));
    HANDLE_ERROR(hipMalloc((void **) &devResults, NUM_OF_BLOCKS * sizeof *devResults));

    for (int i = 0; i < 2; ++i) {
        generateInput(vec[i], size);
        HANDLE_ERROR(hipMemcpy(devVec[i], vec[i], size * sizeof *vec[i],hipMemcpyHostToDevice));
    }
    return 0;
}