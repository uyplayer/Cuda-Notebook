//
// Created by uyplayer on 2024-06-17.
//



#include <error.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>


int main() {
    std::cout << "Hello, concurrentKernels !" << std::endl;


    int nkernels = 8;             // number of concurrent kernels
    int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
    int nbytes = nkernels * sizeof(clock_t);  // number of data bytes
    float kernel_time = 10;                   // time the kernel should run in ms
    float elapsed_time;                       // timing variables
    int cuda_device = 0;


    HANDLE_ERROR(hipGetDevice(&cuda_device));
    hipDeviceProp_t deviceProp;
    HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, cuda_device));

    if (!deviceProp.cooperativeLaunch) {
        std::cerr << "Error: device does not support cooperative kernel launch" << std::endl;
        exit(EXIT_FAILURE);
    }

    if (deviceProp.concurrentKernels == 0) {
        std::cerr << "Error: device does not support concurrent kernel execution" << std::endl;
        exit(EXIT_FAILURE);
    }


    std::cout << "CUDA device " << cuda_device << " : " << deviceProp.name << std::endl;
    std::cout << "Detected Compute SM " << deviceProp.major << "." << deviceProp.minor
              << " hardware with " << deviceProp.multiProcessorCount << " multi-processors";

    clock_t *a = nullptr;
    HANDLE_ERROR(hipHostMalloc((void **) &a, nbytes));

    clock_t *d_a = nullptr;
    HANDLE_ERROR(hipHostMalloc((void **) &d_a, nbytes));

    hipStream_t *streams = new hipStream_t[nstreams];
    for (int i = 0; i < nstreams; i++) {
        HANDLE_ERROR(hipStreamCreate(&(streams[i])));
    }
    hipEvent_t start_event, stop_event;
    HANDLE_ERROR(hipEventCreate(&start_event));
    HANDLE_ERROR(hipEventCreate(&stop_event));





    exit(EXIT_SUCCESS);
}