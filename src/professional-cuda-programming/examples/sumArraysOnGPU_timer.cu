#include "hip/hip_runtime.h"
/*
 * @Author: uyplayer
 * @Date: 2024/4/29 19:53
 * @Email: uyplayer@qq.com
 * @File: sumArraysOnGPU_timer.cu
 * @Software: CLion
 * @Dir: Cuda-Notebook / src/professional-cuda-programming/examples
 * @Project_Name: Cuda-Notebook
 * @Description:
 */


#include <iostream>
#include <hip/hip_runtime.h>
#include "error.h"
#include <GpuTimer.h>
#include <Initializer.h>


__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}


void sumArraysOnHost(float *A, float *B, float *C, const int N) {

    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }

}


void sumArraysOnGPU_timer() {

    std::cout << "Sum Arrays On GPU Timer" << std::endl;


    int dev{0};
    hipDeviceProp_t deviceProp{};
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    std::cout << "Using Device " << dev << ": " << deviceProp.name << std::endl;
    CHECK(hipSetDevice(dev));

    int nElem = 1 << 24;
    std::cout << "Vector size " << nElem << std::endl;

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);


    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = new float[nElem];
    h_B = new float[nElem];

    hostRef = (float *) malloc(nBytes);
    gpuRef = (float *) malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = seconds();

    initialData(h_A, nElem);
    initialData(h_B, nElem);
    iElaps = seconds() - iStart;
    std::cout << "initialData Time elapsed " << iElaps << " sec" << std::endl;
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    iStart = seconds();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = seconds() - iStart;
    std::cout << "sumArraysOnHost Time elapsed " << iElaps << " sec" << std::endl;

    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **) &d_A, nBytes));
    CHECK(hipMalloc((float **) &d_B, nBytes));
    CHECK(hipMalloc((float **) &d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));


    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    iStart = seconds();
    // kernel
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);

    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumArraysOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", grid.x,
           block.x, iElaps);

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

}