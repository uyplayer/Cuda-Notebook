#include "hip/hip_runtime.h"
//
// Created by uyplayer on 2024/5/17.
//





struct hipComplex {

    float r;
    float i;

    __device__ hipComplex(float a, float b) : r(a), i(b) {}

    __device__ float magnitude2(void) {
        return r * r + i * i;
    }

    __device__ hipComplex operator*(const hipComplex &a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    __device__ hipComplex operator+(const hipComplex &a) {
        return hipComplex(r + a.r, i + a.i);
    }


};


__device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}


__global__ void kernel( unsigned char *ptr ) {

    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}
