#include "hip/hip_runtime.h"


//
// Created by uyplayer on 2024/4/28.
//


#include <iostream>
#include "error.h"



void __global__ add2gpu(const double *x, const double *y, double *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}



void add2gpu(){
    std::cout << "Hello, add2gpu!" << std::endl;

    constexpr int N = 1 << 20;
    size_t size = N * sizeof(double);

    double *h_x, *h_y, *h_z;


    h_x = new double[N];
    h_y = new double[N];
    h_z = new double[N];

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = rand() % 100;
        h_y[n] = rand() % 100;
    }

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc(&d_x, size));
    CHECK(hipMalloc(&d_y, size));
    CHECK(hipMalloc(&d_z, size));


    hipEvent_t  start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    CHECK(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice));

    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    add2gpu<<<grid_size, block_size>>>(d_x, d_y, d_z, N);


    CHECK(hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost));

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

    std::cout << "h_z: " << h_z << std::endl;

    std::cout << "Elapsed time: " << elapsed_time << "ms" << std::endl;

    // free
    delete[]  h_x,h_y,h_z;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}