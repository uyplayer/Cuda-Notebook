#include "hip/hip_runtime.h"


//
// Created by uyplayer on 2024/4/29.
//


#include "error.h"


__global__ void printThreadIndex(int *A, const int nx, const int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,ix, iy, idx, A[idx]);

}

void checkThreadIndex() {

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);


    int *h_A;
    h_A = (int *)malloc(nBytes);


    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = i;
    }


    int *d_MatA;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));

    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invoke the kernel
    printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    CHECK(hipFree(d_MatA));
    free(h_A);

    // reset device
    CHECK(hipDeviceReset());


}