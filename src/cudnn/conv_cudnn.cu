//
// Created by uyplayer on 2024-06-06.
//


#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include "error.h"


int main() {
    hipdnnHandle_t cudnn;
    HANDLE_ERROR_CUDNN(hipdnnCreate(&cudnn));

    // input
    constexpr int in_n = 1;
    constexpr int inc_c = 1;
    constexpr int in_h = 5;
    constexpr int in_w = 5;

    std::cout << "in_n: " << in_n << std::endl;
    std::cout << "inc_c: " << inc_c << std::endl;
    std::cout << "in_h: " << in_h << std::endl;
    std::cout << "in_w: " << in_w << std::endl;

    hipdnnTensorDescriptor_t in_desc;
    HANDLE_ERROR_CUDNN(hipdnnCreateTensorDescriptor(&in_desc));
HANDLE_ERROR_CUDNN(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, in_n, inc_c, in_h, in_w));



}


































