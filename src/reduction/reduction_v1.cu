#include "hip/hip_runtime.h"
//
// Created by uyplayer on 2024-05-27.
//


#include <iostream>
#include <../common/error.h>
#include <hip/hip_runtime.h>


__global__ void v1(const float* x, float* y)
{
    __shared__ float sdata[256];
    int tid = threadIdx.x;
    sdata[tid] = x[tid];
    __syncthreads();

    if (tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();


    if (tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    if (tid < 32) sdata[tid] += sdata[tid + 32];
    __syncthreads();

    if (tid < 16) sdata[tid] += sdata[tid + 16];
    __syncthreads();

    if (tid < 8) sdata[tid] += sdata[tid + 8];
    __syncthreads();

    if (tid < 4) sdata[tid] += sdata[tid + 4];
    __syncthreads();

    if (tid < 2) sdata[tid] += sdata[tid + 2];
    __syncthreads();

    if (tid == 0)
    {
        *y = sdata[0] + sdata[1];
    }
}

void reduction_v1()
{
    std::cout << "Hallo Reduction v1" << std::endl;

    constexpr int n_size = 256;
    int byte_size = n_size * sizeof(float);

    float *d_x, *d_y, *h_x;
    HANDLE_ERROR(hipMalloc(&d_x, byte_size));
    HANDLE_ERROR(hipMalloc(&d_y, sizeof(float)));
    HANDLE_ERROR(hipHostMalloc(&h_x, byte_size));

    for (int i = 0; i < n_size; i++)
    {
        h_x[i] = 1.0f;
    }

    hipMemcpy(d_x, h_x, byte_size, hipMemcpyHostToDevice);
    v1<<<1, n_size>>>(d_x, d_y);
    hipDeviceSynchronize();
    float result = 0;
    hipMemcpy(&result, d_y, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(h_x);
}
