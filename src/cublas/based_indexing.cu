//
// Created by uyplayer on 2024/5/20.
//


#include "error.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>

//  Fortran风格的1-based索引和列主存储
// row ,col ,leading dimension(number of rows
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
// C风格的0-based索引和行主存储
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// row and col
#define M 6
#define N 5


// cuBLAS: 0-based indexing
static __inline__ void modify0(hipblasHandle_t handle, float* m, int ldm, int n, int p, int q, float alpha, float beta)
{
    hipblasSscal(handle, n - q + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
    hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}


// cuBLAS: 1-based indexing
void base_1()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;


    float* devPtrA;
    float* a = 0;
    a = (float*)malloc(M * N * sizeof (*a));
    if (a == NULL)
    {
        std::cout << "host memory allocation failed" << std::endl;
        exit(1);
    }
    for (int j = 1; j <= N; j++)
    {
        for (int i = 1; i <= M; i++)
        {
            a[IDX2F(i, j, M)] = (float)(i + (j - 1) * M);
        }
    }

    cudaStat = hipMalloc((void**)&devPtrA, M * N * sizeof(*a));
    if (cudaStat != hipSuccess)
    {
        std::cout << "device memory allocation failed" << std::endl;
        free(a);
        exit(1);
    }
    // initialize handle
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed" << std::endl;
        hipFree(devPtrA);
        free(a);
        exit(1);
    }
    // copy the array to the device
    stat = hipblasSetMatrix(M,N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "data download failed" << std::endl;
        free(a);
        hipFree(devPtrA);
        hipblasDestroy(handle);
        exit(1);
    }

    modify0(handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
    stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "data download failed" << std::endl;
        free(a);
        hipFree(devPtrA);
        hipblasDestroy(handle);
        exit(1);
    }
    hipFree(devPtrA);
    hipblasDestroy(handle);
    for (int j = 1; j <= N; j++)
    {
        for (int i = 1; i <= M; i++)
        {
            std::cout << std::setw(7) << std::setprecision(0) << a[IDX2F(i,j,M)];
        }
        std::cout << std::endl;
    }
    free(a);
}

// cuBLAS: 0-based indexing
void base_0()
{
}
